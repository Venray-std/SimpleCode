#include <stdio.h>
#include <hip/hip_runtime.h> 
#include <iostream>
#include <chrono>
#include <cstdlib>
using std::chrono::high_resolution_clock;
using std::chrono::microseconds;

__global__ void saxpy_par(int n, float a, float *x, float *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

float saxpy_check(int n, float a, float *x, float *y, float *z, int niter)
{
// a, x, y == original data for saxpy
// z = result found -- with which to compare.
    float s=0.0, t = 0.0;
    for (int i=0; i<n; i++) {
        for (int j = 0; j < niter; j++)     // loop NITER in cpu
            y[i] += a * x[i] ;
        s += (y[i] - z[i])*(y[i] - z[i]);   // loss
        t += z[i]*z[i];
    }
    if (t == 0.0) return(-1);
    else
        return(sqrt(s/t));    // rate of loss
}

int main(void)
{
  int N = 1<<23;
  float a = 2.0f;
  float *x, *y,*z, *d_x, *d_y;
  x = (float*)malloc(N*sizeof(float));
  y = (float*)malloc(N*sizeof(float));
  z = (float*)malloc(N*sizeof(float));
  hipMalloc(&d_x, N*sizeof(float)); 
  hipMalloc(&d_y, N*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = rand();
    y[i] = rand();
  }
  int NITER = 100;

  
  for (int vecLen = 2048; vecLen <= N; vecLen*=2){
    long opNum =256*((vecLen + 255) / 256)*2;   // operator num, 256*((vecLen + 255) / 256) threads and one thread with two operators
    hipMemcpy(d_x, x, vecLen*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, vecLen*sizeof(float), hipMemcpyHostToDevice);
    
    high_resolution_clock::time_point beginTime = high_resolution_clock::now();
    for (int iter = 0 ;iter<NITER; iter++){
    // data transfers included in timing       
    // Perform SAXPY on 1M elements
        saxpy_par<<<(vecLen+255)/256, 256>>>(vecLen, a, d_x, d_y);  // loop NITER
    }
    high_resolution_clock::time_point endTime = high_resolution_clock::now();   // end time
    hipMemcpy(z, d_y, vecLen*sizeof(float), hipMemcpyDeviceToHost);   //  copy data from gpu to cpu
    microseconds timeInterval = std::chrono::duration_cast<microseconds>(endTime - beginTime);  // Time difference,unit is microseconds
    double time2 = timeInterval.count();  
    time2 /= NITER;
    double MFLOPS = opNum/time2;    // MFLOPS, one second number of operators
    double error = saxpy_check(vecLen, a, x, y, z, NITER);   // check
    printf("vecLen = %ld, Mflops = %f, Error = %f\n", vecLen,MFLOPS, error);
  }
  hipFree(d_x);
  hipFree(d_y);
  free(x);
  free(y);
  free(z);
}